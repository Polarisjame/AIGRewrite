#include "hip/hip_runtime.h"
#include "src/abc_patch_gpu_test.h"

extern "C" int Abc_CommandBalance( Abc_Frame_t * pAbc, int argc, char ** argv );
extern "C" int Abc_CommandRewrite( Abc_Frame_t * pAbc, int argc, char ** argv );
extern "C" int Abc_CommandRefactor( Abc_Frame_t * pAbc, int argc, char ** argv );

__global__ void printNodesKernel(const int * pnNodes) {
    std::printf("AIG stats: nNodes = %d", *pnNodes);
}

int printNodes(AIGMan* gpuMan) {
    if(gpuMan->deviceAllocated){
        printNodesKernel<<<1,1>>>(gpuMan->d_pnNodes);
    }
    else{
        std::printf("%d\n",gpuMan->nNodes);
    }
    return 1;
}

int getNodesGpu(AIGMan* gpuMan){
    int nNodes;

    if(gpuMan->deviceAllocated){
        hipMemcpy(&nNodes, gpuMan->d_pnNodes, sizeof(int), hipMemcpyDeviceToHost);
    }
    else{
        nNodes = gpuMan->nNodes;
    }

    return nNodes;
}

int getNodesAbc(Abc_Frame_t* pAbc){
    Abc_Ntk_t* pNtk = pAbc->pNtkCur;
    return pNtk->vObjs->nSize - pNtk->vCis->nSize - pNtk->vCos->nSize - 1;
}

bool isStable(int currNodes, int prevNodes){
    int delta = prevNodes-currNodes;
    if(delta<0) return false;
    if((float)delta < 200) return true;
    // if (currNodes==prevNodes) return true;
    return false;
} 

int AIGRefresh(AIGMan* gpuMan){
    gpuMan->toHost();
    gpuMan->clearDevice();
    hipDeviceSynchronize();
    return 1;
}


void Raina_balance(Abc_Frame_t * pAbc)
{
    char * balance_argv[1];
    balance_argv[0] = strdup("balance");
    int ret_balance = Abc_CommandBalance( pAbc, 1, balance_argv );
    free(balance_argv[0]);
    if(ret_balance != 0){
        Abc_Print(-1, "balance failed with code %d.\n",ret_balance);
        return;
    }
}

//rewrite (rw)
void Raina_rewirte(Abc_Frame_t * pAbc)
{
    char * rewrite_argv[1];
    rewrite_argv[0] = strdup("rewrite");
    int ret_rewrite = Abc_CommandRewrite( pAbc, 1, rewrite_argv);
    free(rewrite_argv[0]);
    if(ret_rewrite != 0){
        Abc_Print(-1, "rw failed with code %d.\n", ret_rewrite);
        return;
    }
}

//rewrite (rwz)
void Raina_rewritez(Abc_Frame_t * pAbc)
{
    char * rewritez_argv[2];
    rewritez_argv[0] = strdup("rewrite");
    rewritez_argv[1] = strdup("-z");
    int ret_rewritez = Abc_CommandRewrite( pAbc, 2, rewritez_argv);
    free(rewritez_argv[0]);
    free(rewritez_argv[1]);
    if(ret_rewritez != 0){
        Abc_Print(-1, "rwz failed with code %d.\n", ret_rewritez);
        return;
    }
}

//refactor (rf)
void Raina_refactor(Abc_Frame_t * pAbc)
{
    char * refactor_argv[1];
    refactor_argv[0] = strdup("refactor");
    int ret_refactor = Abc_CommandRefactor( pAbc, 1, refactor_argv);
    free(refactor_argv[0]);
    if(ret_refactor != 0){
        Abc_Print(-1, "refactor failed with code %d.\n", ret_refactor);
        return;
    }
}

//refactor (rfz)
void Raina_refactorz(Abc_Frame_t * pAbc)
{
    char * refactorz_argv[2];
    refactorz_argv[0] = strdup("refactor");
    refactorz_argv[1] = strdup("-z");
    int ret_refactorz = Abc_CommandRefactor( pAbc, 2, refactorz_argv);
    free(refactorz_argv[0]);
    free(refactorz_argv[1]);
    if(ret_refactorz != 0){
        Abc_Print(-1, "refactor failed with code %d.\n", ret_refactorz);
        return;
    }
}


/*
alias resyn2      "b; rw; rf; b; rw; rwz; b; rfz; rwz; b"
*/
void Raina_resyn2(Abc_Frame_t * pAbc)
{
    //b
    Raina_balance(pAbc);
    //rw
    Raina_rewirte(pAbc);
    //rf
    Raina_refactor(pAbc);
    //b
    Raina_balance(pAbc);
    //rw
    Raina_rewirte(pAbc);
    //rwz
    Raina_rewritez(pAbc);
    //b
    Raina_balance(pAbc);
    //rfz
    Raina_refactorz(pAbc);
    //rwz
    Raina_rewritez(pAbc);
    //b
    Raina_balance(pAbc);
}

int testFlow(std::string fpath){
    Abc_Frame_t * pAbc = Abc_FrameGetGlobalFrame();
    // hipMalloc(&d_pnNodes, sizeof(int));
    Raina_CommandGpuRead(pAbc, fpath);
    // Raina_CommandGpuPrintStats(pAbc);

    AIGMan* gpuMan = getGpuMan();

    Abc_Ntk_t * pNtkNew = GpuManToAbcNtk(gpuMan);
    Abc_FrameReplaceCurrentNetwork(pAbc, pNtkNew);

    bool fUpdateLevel = true;
    int rfCutSize = 8;

    int iterCount = 0;
    // int prevNodes = getNodesGpu(gpuMan);
    int prevNodes = getNodesAbc(pAbc);
    int stableRounds = 0;
    std::printf("oriNodes:%d\n",prevNodes);
    
    while(1){
        iterCount++;
        // ***** GPUResyn2
        // Raina_CommandGpuResyn2(pAbc, rfCutSize, fUpdateLevel);
        // hipDeviceSynchronize();
        // int currNodes = getNodes(gpuMan);
        
        // ***** AbcResyn2
        Raina_resyn2(pAbc);
        int currNodes = getNodesAbc(pAbc);
        std::printf("Iter:%d currNodes:%d\n",iterCount, currNodes);
        if(isStable(currNodes, prevNodes)){
            stableRounds++;
            while(stableRounds < 3){
                iterCount++;
                // Raina_CommandGpuResyn2(pAbc, rfCutSize, fUpdateLevel);
                // hipDeviceSynchronize();
                // int currNodes = getNodesGpu(gpuMan);
                Raina_resyn2(pAbc);
                int currNodes = getNodesAbc(pAbc);
                std::printf("Iter:%d currNodes:%d\n",iterCount, currNodes);
                if(!isStable(currNodes, prevNodes)){
                    stableRounds=0;
                    break;
                }
                stableRounds++;
            }
        }
        if (stableRounds==3) break;
        prevNodes = currNodes;
    }

    return 1;
}